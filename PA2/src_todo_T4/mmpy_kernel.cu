#include "hip/hip_runtime.h"
// ;-*- mode: c;-*-
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include "../src/utils.h"
#include "../src/types.h"
#include "mytypes.h"
using namespace std;

#include <stdio.h>

#if defined(NAIVE)
__global__ void matMul(int N, _FTYPE_ *C, _FTYPE_ *A, _FTYPE_ *B) {

    int I =  blockIdx.y*blockDim.y + threadIdx.y;
    int J =  blockIdx.x*blockDim.x + threadIdx.x;

    if((I < N) && (J < N)){
        _FTYPE_ _c = 0;
        for (unsigned int k = 0; k < N; k++) {
            _FTYPE_ a = A[I * N + k];
            _FTYPE_ b = B[k * N + J];
            _c += a * b;
        }
        C[I * N + J] = _c;
    }
}

#elif defined(SMEM)
//You should be changing the kernel here for the non naive implementation.
__global__ void matMul(int N, _FTYPE_ *C, _FTYPE_ *A, _FTYPE_ *B) {

    extern __shared__ _FTYPE_ sMem[];
    _FTYPE_ (*As)[TILEDIM_K] = (_FTYPE_ (*)[TILEDIM_K])sMem;
    _FTYPE_ (*Bs)[TILEDIM_N] = (_FTYPE_ (*)[TILEDIM_N])&sMem[TILEDIM_M * TILEDIM_K];

    int ty = threadIdx.y;
    int tx = threadIdx.x;
    int by = blockIdx.y;
    int bx = blockIdx.x;

    int I =  by*TILEDIM_M + ty;
    int J =  bx*TILEDIM_N + tx;

    float Cij = 0.0;

    for (int kk = 0; kk < N; kk+=TILEDIM_K)
    {
        if (I < N && kk+tx < N){
            As[ty][tx] = A[I*N + kk + tx];
        }else{
            As[ty][tx] = 0;
        }

        if (J < N && kk+ty < N){
            Bs[ty][tx] = B[(kk + ty)*N + J];
        }else{
            Bs[ty][tx] = 0;
        }

        __syncthreads();

        for (int k = 0; k < TILEDIM_K; k++)
        {
            /* code */
            Cij += As[ty][k] * Bs[k][tx];
        }
        __syncthreads();
    }

    if (I < N && J < N)
    {
        C[I*N + J] = Cij;
    }
    
}

#else
//You should be changing the kernel here for the non naive implementation.
__global__ void matMul(int N, _FTYPE_ *C, _FTYPE_ *A, _FTYPE_ *B) {

    extern __shared__ _FTYPE_ sMem[];
    _FTYPE_ (*As) = (_FTYPE_ (*))sMem;
    _FTYPE_ (*Bs) = (_FTYPE_ (*))&sMem[TILEDIM_M * TILEDIM_K];

    const uint ty = threadIdx.y;
    const uint tx = threadIdx.x;
    const uint by = blockIdx.y;
    const uint bx = blockIdx.x;

    _FTYPE_ Cijs[TILESCALE_M * TILESCALE_N] = {0.0};
    _FTYPE_ A_buf[TILESCALE_M] = {0.0};
    _FTYPE_ B_buf[TILESCALE_N] = {0.0};

    // assign (by, bx) threads to load values into As (T_M, T_K)
    const uint Ay = (ty * blockDim.x + tx) / TILEDIM_K;
    const uint Ax = (ty * blockDim.x + tx) % TILEDIM_K;
    // assign (by, bx) threads to load values into Bs (T_K, T_N)
    const uint By = (ty * blockDim.x + tx) / TILEDIM_N;
    const uint Bx = (ty * blockDim.x + tx) % TILEDIM_N;

    const uint As_offset = (blockDim.x * blockDim.y) / TILEDIM_K;
    const uint Bs_offset = (blockDim.x * blockDim.y) / TILEDIM_N;

    #pragma unroll
    for (uint kk = 0; kk < N; kk+=TILEDIM_K){
        #pragma unroll
        for (uint i = 0; i < TILEDIM_M; i+=As_offset){
            if (((by * TILEDIM_M + Ay+i) < N) && ((kk + Ax) < N)){
                As[(Ay + i) * TILEDIM_K + Ax] = A[(by * TILEDIM_M + Ay + i) * N + kk + Ax];
            }else{
                As[(Ay + i) * TILEDIM_K + Ax] = 0;
            }
        }
        
        #pragma unroll
        for (uint i = 0; i < TILEDIM_K; i+=Bs_offset){
            if (((kk + By + i) < N) && ((bx*TILEDIM_N + Bx) < N)){
                Bs[(By + i) * TILEDIM_N + Bx] = B[(kk + By + i) * N + bx*TILEDIM_N + Bx];
            }else{
                Bs[(By + i) * TILEDIM_N + Bx] = 0;
            }
        }

        __syncthreads();

        #pragma unroll
        for (uint k = 0; k < TILEDIM_K; ++k)
        {
            #pragma unroll
            for (uint i = 0; i < TILESCALE_M; ++i){
                // seperate
                //A_buf[i] = As[(ty + (i * blockDim.y)) * TILEDIM_K + k];
                // adjacent
                A_buf[i] = As[((ty * TILESCALE_M) + i) * TILEDIM_K + k];
            }
            #pragma unroll
            for (uint i = 0; i < TILESCALE_N; ++i){
                // seperate
                //B_buf[i] = Bs[(k * TILEDIM_N) + tx + (i * blockDim.x)];
                // adjacent
                B_buf[i] = Bs[(k * TILEDIM_N) + (tx * TILESCALE_N) + i];
            }
            
            #pragma unroll
            for (uint i = 0; i < TILESCALE_M; ++i){
                #pragma unroll
                for (uint j = 0; j < TILESCALE_N; ++j){
                    Cijs[i * TILESCALE_M + j] += A_buf[i] * B_buf[j];
                }
            }
        }
        __syncthreads();
    }

    #pragma unroll
    for (uint i = 0; i < TILESCALE_M; ++i){
        #pragma unroll
        for (uint j = 0; j < TILESCALE_N; ++j){
            // seperate
            //if ((by*TILEDIM_M + ty + (i * blockDim.y)) < N && (bx*TILEDIM_N + tx + (j * blockDim.x)) < N){
            //    C[(by*TILEDIM_M + ty + (i * blockDim.y)) * N + (bx*TILEDIM_N + tx + (j * blockDim.x))] = Cijs[i * TILESCALE_M + j];
            //}
            // adjacent
            if ((by*TILEDIM_M + (ty * TILESCALE_M) + i) < N && (bx*TILEDIM_N + (tx * TILESCALE_N) + j) < N){
                C[(by*TILEDIM_M + (ty * TILESCALE_M) + i) * N + (bx*TILEDIM_N + (tx * TILESCALE_N) + j)] = Cijs[i * TILESCALE_M + j];
            }
        }
    }
}
#endif